#include "kernels.cuh"
#include "host_functions.cuh"

int main(int argc, char* argv[]) {
    if (argc < 3) {
        std::cerr << "Error: Usage: " << argv[0] << " <velocity set> <ID>" << std::endl;
        return 1;
    }
    std::string VELOCITY_SET = argv[1];
    std::string SIM_ID = argv[2];

    std::string SIM_DIR = createSimulationDirectory(VELOCITY_SET,SIM_ID);
    //computeAndPrintOccupancy();
    initDeviceVars();

    // ================================================================================================== //

    dim3 threadsPerBlock(BLOCK_SIZE_X,BLOCK_SIZE_Y,BLOCK_SIZE_Z);
    dim3 numBlocks((NX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (NY + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (NZ + threadsPerBlock.z - 1) / threadsPerBlock.z);
                        
    dim3 threadsPerBlockX(BLOCK_SIZE_Y*2,BLOCK_SIZE_Z*2);
    dim3 numBlocksX((NY + threadsPerBlockX.x - 1) / threadsPerBlockX.x,
                    (NZ + threadsPerBlockX.y - 1) / threadsPerBlockX.y);

    dim3 threadsPerBlockY(BLOCK_SIZE_X*2,BLOCK_SIZE_Z*2);
    dim3 numBlocksY((NX + threadsPerBlockY.x - 1) / threadsPerBlockY.x,
                    (NZ + threadsPerBlockY.y - 1) / threadsPerBlockY.y);

    dim3 threadsPerBlockZ(BLOCK_SIZE_X*2,BLOCK_SIZE_Y*2);  
    dim3 numBlocksZ((NX + threadsPerBlockZ.x - 1) / threadsPerBlockZ.x,
                    (NY + threadsPerBlockZ.y - 1) / threadsPerBlockZ.y);
                    
    hipStream_t mainStream;
    checkCudaErrors(hipStreamCreate(&mainStream));

    gpuInitFieldsAndDistributions<<<numBlocks,threadsPerBlock,0,mainStream>>> (lbm); 
    getLastCudaError("gpuInitFieldsAndDistributions");

    auto START_TIME = std::chrono::high_resolution_clock::now();
    for (int STEP = 0; STEP <= NSTEPS ; ++STEP) {
        std::cout << "Passo " << STEP << " de " << NSTEPS << " iniciado..." << std::endl;

        // =================================== INFLOW =================================== //

            gpuApplyInflow<<<numBlocksZ,threadsPerBlockZ,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm,STEP); 
            getLastCudaError("gpuApplyInflow");
            gpuApplyLatInflow<<<numBlocksY,threadsPerBlockY,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm); 
            getLastCudaError("gpuApplyLatInflow");

        // =============================================================================  //
        
        // ========================= COLLISION & STREAMING ========================= //
            
            gpuEvolvePhaseField<<<numBlocks,threadsPerBlock,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm); 
            getLastCudaError("gpuEvolvePhaseField");
            gpuMomCollisionStream<<<numBlocks,threadsPerBlock,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm); 
            getLastCudaError("gpuMomCollisionStream");

        // ========================================================================= //    

        // =================================== BOUNDARIES =================================== //

            gpuReconstructBoundaries<<<numBlocks,threadsPerBlock,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm); 
            getLastCudaError("gpuReconstructBoundaries");
            gpuApplyOutflowX<<<numBlocksX,threadsPerBlockX,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm);
            getLastCudaError("gpuApplyOutflowZ");
            gpuApplyOutflowY<<<numBlocksY,threadsPerBlockY,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm);
            getLastCudaError("gpuApplyOutflowZ");
            gpuApplyOutflowZ<<<numBlocksZ,threadsPerBlockZ,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm);
            getLastCudaError("gpuApplyOutflowZ");

        // ================================================================================== //

        // =================================== DERIVED FIELDS =================================== //

            //gpuDerivedFields<<<numBlocks,threadsPerBlock,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm,dfields); 
            //getLastCudaError("gpuDerivedFields");

        // ====================================================================================== //

        checkCudaErrors(hipDeviceSynchronize());

        if (STEP % MACRO_SAVE == 0) {

            copyAndSaveToBinary(lbm.phi,NX*NY*NZ,SIM_DIR,SIM_ID,STEP,"phi");
            copyAndSaveToBinary(lbm.rho,NX*NY*NZ,SIM_DIR,SIM_ID,STEP,"rho");
            copyAndSaveToBinary(lbm.uy,NX*NY*NZ,SIM_DIR,SIM_ID,STEP,"uy");
            copyAndSaveToBinary(lbm.uz,NX*NY*NZ,SIM_DIR,SIM_ID,STEP,"uz");
            //copyAndSaveToBinary(dfields.vorticity_mag,NX*NY*NZ,SIM_DIR,SIM_ID,STEP,"vorticity_mag");
            //copyAndSaveToBinary(dfields.q_criterion,NX*NY*NZ,SIM_DIR,SIM_ID,STEP,"q_criterion");

            std::cout << "Passo " << STEP << ": Dados salvos em " << SIM_DIR << std::endl;
        }
    }
    auto END_TIME = std::chrono::high_resolution_clock::now();

    checkCudaErrors(hipStreamDestroy(mainStream));

    // lbmfields
    hipFree(lbm.f); 
    hipFree(lbm.g);
    hipFree(lbm.phi); 
    hipFree(lbm.rho);
    hipFree(lbm.normx);
    hipFree(lbm.normy); 
    hipFree(lbm.normz);
    hipFree(lbm.ux); 
    hipFree(lbm.uy); 
    hipFree(lbm.uz);
    hipFree(lbm.pxx);
    hipFree(lbm.pyy);
    hipFree(lbm.pzz);
    hipFree(lbm.pxy);
    hipFree(lbm.pxz);
    hipFree(lbm.pyz);
    hipFree(lbm.ffx); 
    hipFree(lbm.ffy); 
    hipFree(lbm.ffz);

    // derivedfields
    hipFree(dfields.vorticity_mag);
    hipFree(dfields.q_criterion);

    std::chrono::duration<double> ELAPSED_TIME = END_TIME - START_TIME;
    long long TOTAL_CELLS = static_cast<long long>(NX) * NY * NZ * NSTEPS;
    double MLUPS = static_cast<double>(TOTAL_CELLS) / (ELAPSED_TIME.count() * 1e6);

    std::cout << "\n// =============================================== //\n";
    std::cout << "     Total execution time    : " << ELAPSED_TIME.count() << " seconds\n";
    std::cout << "     Performance             : " << MLUPS << " MLUPS\n";
    std::cout << "// =============================================== //\n" << std::endl;

    generateSimulationInfoFile(SIM_DIR,SIM_ID,VELOCITY_SET,NSTEPS,MACRO_SAVE,TAU,MLUPS);
    getLastCudaError("Final sync");
    return 0;
}
