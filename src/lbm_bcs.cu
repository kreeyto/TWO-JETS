#include "hip/hip_runtime.h"
#include "kernels.cuh"

#define INFLOW_CASE_THREE

__global__ void gpuApplyInflow(LBMFields d, const int STEP) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = 0;

    if (x >= NX || y >= NY) return;

    float center_x = (NX-1) * 0.5f;
    float center_y = (NY-1) * 0.5f;

    float dx = x-center_x, dy = y-center_y;
    float radial_dist = sqrtf(dx*dx + dy*dy);
    float radius = 0.5f * DIAM;
    if (radial_dist > radius) return;

    #ifdef INFLOW_CASE_ONE 
        float radial_dist_norm = radial_dist / radius;
        float envelope = 1.0f - gpu_smoothstep(0.6f, 1.0f, radial_dist_norm);
        float profile = 0.5f + 0.5f * tanhf(2.0f * (radius - radial_dist) / 3.0f);
        float phi_in = profile * envelope; 
        #ifdef PERTURBATION
            float uz_in = U_JET * (1.0f + DATAZ[STEP/MACRO_SAVE] * 10.0f) * phi_in;
        #else
            float uz_in = U_JET * phi_in;
        #endif
    #elif defined(INFLOW_CASE_TWO)
        float radial_dist_norm = radial_dist / radius;
        float envelope = 1.0f - gpu_smoothstep(0.6f, 1.0f, radial_dist_norm);
        float phi_in = 1.0f;
        #ifdef PERTURBATION
            float uz_in = U_JET * (1.0f + DATAZ[STEP/MACRO_SAVE] * 10.0f) * envelope;
        #else
            float uz_in = U_JET * envelope;
        #endif
    #elif defined(INFLOW_CASE_THREE) 
        float phi_in = 1.0f;
        #ifdef PERTURBATION
            float uz_in = U_JET * (1.0f + DATAZ[STEP/MACRO_SAVE] * 10.0f);
        #else
            float uz_in = U_JET
        #endif
    #endif

    float rho_val = 1.0f;
    float uu = 1.5f * (uz_in * uz_in);

    const idx_t idx3_in = gpu_idx_global3(x,y,z);
    d.rho[idx3_in] = rho_val;
    d.phi[idx3_in] = phi_in;
    d.ux[idx3_in] = 0.0f;
    d.uy[idx3_in] = 0.0f;
    d.uz[idx3_in] = uz_in;

    #pragma unroll FLINKS
    for (int Q = 0; Q < FLINKS; ++Q) {
        const int xx = x + CIX[Q];
        const int yy = y + CIY[Q];
        const int zz = z + CIZ[Q];
        float feq = gpu_compute_equilibria(rho_val,0.0f,0.0f,uz_in,uu,Q);
        const idx_t streamed_idx4 = gpu_idx_global4(xx,yy,zz,Q);
        d.f[streamed_idx4] = to_dtype(feq);
    }
    #pragma unroll GLINKS
    for (int Q = 0; Q < GLINKS; ++Q) {
        const int xx = x + CIX[Q];
        const int yy = y + CIY[Q];
        const int zz = z + CIZ[Q];
        float geq = gpu_compute_truncated_equilibria(phi_in,0.0f,0.0f,uz_in,Q);
        const idx_t streamed_idx4 = gpu_idx_global4(xx,yy,zz,Q);
        d.g[streamed_idx4] = geq;
    }
}

__global__ void gpuApplyLatInflow(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;  
    const int y = 0;
    const int z = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= NX || z >= NZ) return;

    float center_x = (NX-1) * 0.5f;
    float center_z = (NZ-1) * 0.5f;

    float dx = x-center_x, dz = z-center_z;
    float radial_dist = sqrtf(dx*dx + dz*dz);
    float radius = 0.5f * DIAM;

    if (radial_dist > radius) return;

    float phi_in = 0.0f;               
    float uy_in = U_JET;              
    float rho_val = 1.0f;
    float uu = 1.5f * (uy_in * uy_in);

    const idx_t idx3_in = gpu_idx_global3(x,y,z);
    d.rho[idx3_in] = rho_val;
    d.phi[idx3_in] = phi_in;
    d.ux[idx3_in] = 0.0f;
    d.uy[idx3_in] = uy_in;
    d.uz[idx3_in] = 0.0f;

    #pragma unroll FLINKS
    for (int Q = 0; Q < FLINKS; ++Q) {
        const int xx = x + CIX[Q];
        const int yy = y + CIY[Q];
        const int zz = z + CIZ[Q];
        float feq = gpu_compute_equilibria(rho_val,0.0f,uy_in,0.0f,uu,Q);
        const idx_t streamed_idx4 = gpu_idx_global4(xx,yy,zz,Q);
        d.f[streamed_idx4] = to_dtype(feq);
    }

    #pragma unroll GLINKS
    for (int Q = 0; Q < GLINKS; ++Q) {
        const int xx = x + CIX[Q];
        const int yy = y + CIY[Q];
        const int zz = z + CIZ[Q];
        float geq = gpu_compute_truncated_equilibria(phi_in,0.0f,uy_in,0.0f,Q);
        const idx_t streamed_idx4 = gpu_idx_global4(xx,yy,zz,Q);
        d.g[streamed_idx4] = geq;
    }
}

__global__ void gpuReconstructBoundaries(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    bool is_valid_edge = (x < NX && y < NY && z < NZ) &&
                         (x == 0 || x == NX-1 || y == NY-1 || z == NZ-1); 
    if (!is_valid_edge) return;
    const idx_t idx3 = gpu_idx_global3(x,y,z);

    #pragma unroll FLINKS
    for (int Q = 0; Q < FLINKS; ++Q) {
        const int xx = x + CIX[Q];
        const int yy = y + CIY[Q];
        const int zz = z + CIZ[Q];
        if (xx >= 0 && xx < NX && yy >= 0 && yy < NY && zz >= 0 && zz < NZ) {
            const idx_t streamed_idx4 = gpu_idx_global4(xx,yy,zz,Q);
            d.f[streamed_idx4] = to_dtype(W[Q] * d.rho[idx3] - W[Q]);
        }
    }
    #pragma unroll GLINKS
    for (int Q = 0; Q < GLINKS; ++Q) {
        const int xx = x + CIX[Q];
        const int yy = y + CIY[Q];
        const int zz = z + CIZ[Q];
        if (xx >= 0 && xx < NX && yy >= 0 && yy < NY && zz >= 0 && zz < NZ) {
            const idx_t streamed_idx4 = gpu_idx_global4(xx,yy,zz,Q);
            d.g[streamed_idx4] = W_G[Q] * d.phi[idx3] - W_G[Q];
        }
    }
}

// ============================================================================================================== //

__global__ void gpuApplyOutflowX(LBMFields d) {
    const int x = NX-1;
    const int y = threadIdx.x + blockIdx.x * blockDim.x;
    const int z = threadIdx.y + blockIdx.y * blockDim.y;

    if (y >= NY || z >= NZ) return;

    d.phi[gpu_idx_global3(x,y,z)] = d.phi[gpu_idx_global3(x-1,y,z)];
    d.rho[gpu_idx_global3(x,y,z)] = d.rho[gpu_idx_global3(x-1,y,z)];
    d.ux[gpu_idx_global3(x,y,z)] = d.ux[gpu_idx_global3(x-1,y,z)];
    d.uy[gpu_idx_global3(x,y,z)] = d.uy[gpu_idx_global3(x-1,y,z)];
    d.uz[gpu_idx_global3(x,y,z)] = d.uz[gpu_idx_global3(x-1,y,z)];
}

__global__ void gpuApplyOutflowY(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = NY-1;
    const int z = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= NX || z >= NZ) return;

    d.phi[gpu_idx_global3(x,y,z)] = d.phi[gpu_idx_global3(x,y-1,z)];
    d.rho[gpu_idx_global3(x,y,z)] = d.rho[gpu_idx_global3(x,y-1,z)];
    d.ux[gpu_idx_global3(x,y,z)] = d.ux[gpu_idx_global3(x,y-1,z)];
    d.uy[gpu_idx_global3(x,y,z)] = d.uy[gpu_idx_global3(x,y-1,z)];
    d.uz[gpu_idx_global3(x,y,z)] = d.uz[gpu_idx_global3(x,y-1,z)];
}

__global__ void gpuApplyOutflowZ(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = NZ-1;

    if (x >= NX || y >= NY) return;

    d.phi[gpu_idx_global3(x,y,z)] = d.phi[gpu_idx_global3(x,y,z-1)];
    d.rho[gpu_idx_global3(x,y,z)] = d.rho[gpu_idx_global3(x,y,z-1)];
    d.ux[gpu_idx_global3(x,y,z)] = d.ux[gpu_idx_global3(x,y,z-1)];
    d.uy[gpu_idx_global3(x,y,z)] = d.uy[gpu_idx_global3(x,y,z-1)];
    d.uz[gpu_idx_global3(x,y,z)] = d.uz[gpu_idx_global3(x,y,z-1)];
}





