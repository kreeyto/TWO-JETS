#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void gpuPhi(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    const idx_t idx3 = gpu_idx_global3(x,y,z);

    float pop[GLINKS];
    pop[0] = d.g[gpu_idx_global4(x,y,z,0)];
    pop[1] = d.g[gpu_idx_global4(x,y,z,1)];
    pop[2] = d.g[gpu_idx_global4(x,y,z,2)];
    pop[3] = d.g[gpu_idx_global4(x,y,z,3)];
    pop[4] = d.g[gpu_idx_global4(x,y,z,4)];
    pop[5] = d.g[gpu_idx_global4(x,y,z,5)];
    pop[6] = d.g[gpu_idx_global4(x,y,z,6)];

    const float phi_val = (pop[0] + pop[1] + pop[2] + pop[3] + pop[4] + pop[5] + pop[6]) + 1.0f;
        
    d.phi[idx3] = phi_val;
}

__global__ void gpuGradients(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    const idx_t idx3 = gpu_idx_global3(x,y,z);

    const float gradx = 0.375f * (d.phi[gpu_idx_global3(x+1,y,z)] - d.phi[gpu_idx_global3(x-1,y,z)]);
    const float grady = 0.375f * (d.phi[gpu_idx_global3(x,y+1,z)] - d.phi[gpu_idx_global3(x,y-1,z)]);
    const float gradz = 0.375f * (d.phi[gpu_idx_global3(x,y,z+1)] - d.phi[gpu_idx_global3(x,y,z-1)]);

    const float grad2 = gradx*gradx + grady*grady + gradz*gradz;
    const float mag = rsqrtf(grad2 + 1e-6f);
    const float ind_val = grad2 * mag;
    const float normx_val = gradx * mag;
    const float normy_val = grady * mag;
    const float normz_val = gradz * mag;

    d.normx[idx3] = normx_val;
    d.normy[idx3] = normy_val;
    d.normz[idx3] = normz_val;
    d.ind[idx3] = ind_val;
}

__global__ void gpuForces(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    const idx_t idx3 = gpu_idx_global3(x,y,z);

    const float ind_val = d.ind[idx3];
    const float normx_val = d.normx[idx3];
    const float normy_val = d.normy[idx3];
    const float normz_val = d.normz[idx3];

    const float curvature = -0.375f * (d.normx[gpu_idx_global3(x+1,y,z)] - d.normx[gpu_idx_global3(x-1,y,z)] +
                                       d.normy[gpu_idx_global3(x,y+1,z)] - d.normy[gpu_idx_global3(x,y-1,z)] +
                                       d.normz[gpu_idx_global3(x,y,z+1)] - d.normz[gpu_idx_global3(x,y,z-1)]);

    const float coeff_force = SIGMA * curvature;
    d.ffx[idx3] = coeff_force * normx_val * ind_val;
    d.ffy[idx3] = coeff_force * normy_val * ind_val;
    d.ffz[idx3] = coeff_force * normz_val * ind_val;
}

__global__ void gpuEvolvePhaseField(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;
        
    const idx_t idx3 = gpu_idx_global3(x,y,z);

    const float phi_val = d.phi[idx3];
    const float ux_val = d.ux[idx3];
    const float uy_val = d.uy[idx3];
    const float uz_val = d.uz[idx3];
    const float normx_val = d.normx[idx3];
    const float normy_val = d.normy[idx3];
    const float normz_val = d.normz[idx3];
    const float phi_norm = GAMMA * phi_val * (1.0f - phi_val);
    #pragma unroll GLINKS
    for (int Q = 0; Q < GLINKS; ++Q) {
        const int xx = x + CIX[Q];
        const int yy = y + CIY[Q];
        const int zz = z + CIZ[Q];
        const float geq = gpu_compute_truncated_equilibria(phi_val,ux_val,uy_val,uz_val,Q);
        const float anti_diff = W_G[Q] * phi_norm * (CIX[Q] * normx_val + CIY[Q] * normy_val + CIZ[Q] * normz_val);
        const idx_t streamed_idx4 = gpu_idx_global4(xx,yy,zz,Q);
        d.g[streamed_idx4] = geq + anti_diff;
    }
} 
