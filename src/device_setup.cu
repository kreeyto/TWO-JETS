#include "kernels.cuh"

__constant__ float W[FLINKS];
__constant__ float W_G[GLINKS];

__constant__ ci_t CIX[FLINKS], CIY[FLINKS], CIZ[FLINKS];

#ifdef PERTURBATION
    __constant__ float DATAZ[200];
#endif

LBMFields lbm;
DerivedFields dfields;
                                         
// =============================================================================================================================================================== //

void initDeviceVars() {
    size_t SIZE =        NX * NY * NZ          * sizeof(float);            
    size_t F_DIST_SIZE = NX * NY * NZ * FLINKS * sizeof(dtype_t); 
    size_t G_DIST_SIZE = NX * NY * NZ * GLINKS * sizeof(float); 

    checkCudaErrors(hipMalloc(&lbm.phi,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.rho,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.ux,    SIZE));
    checkCudaErrors(hipMalloc(&lbm.uy,    SIZE));
    checkCudaErrors(hipMalloc(&lbm.uz,    SIZE));
    checkCudaErrors(hipMalloc(&lbm.pxx,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.pyy,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.pzz,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.pxy,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.pxz,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.pyz,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.normx, SIZE));
    checkCudaErrors(hipMalloc(&lbm.normy, SIZE));
    checkCudaErrors(hipMalloc(&lbm.normz, SIZE));
    checkCudaErrors(hipMalloc(&lbm.ffx,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.ffy,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.ffz,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.f,     F_DIST_SIZE));
    checkCudaErrors(hipMalloc(&lbm.g,     G_DIST_SIZE));

    checkCudaErrors(hipMalloc(&dfields.vorticity_mag, SIZE));
    checkCudaErrors(hipMalloc(&dfields.q_criterion,   SIZE));

    checkCudaErrors(hipMemset(lbm.phi,   0, SIZE));
    checkCudaErrors(hipMemset(lbm.ux,    0, SIZE));
    checkCudaErrors(hipMemset(lbm.uy,    0, SIZE));
    checkCudaErrors(hipMemset(lbm.uz,    0, SIZE));
    checkCudaErrors(hipMemset(lbm.normx, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.normy, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.normz, 0, SIZE));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(W),   &H_W,   FLINKS * sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(W_G), &H_W_G, GLINKS * sizeof(float)));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CIX),   &H_CIX,   FLINKS * sizeof(ci_t)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CIY),   &H_CIY,   FLINKS * sizeof(ci_t)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CIZ),   &H_CIZ,   FLINKS * sizeof(ci_t)));

    #ifdef PERTURBATION
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(DATAZ), &H_DATAZ, 200 * sizeof(float)));
    #endif

    getLastCudaError("initDeviceVars: post-initialization");
}

